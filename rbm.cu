#include <iostream>
#include <hipblas.h>
#include <hiprand.h>
#include "rbm.h"
#include "rbm_kernels.h"

#define DEBUG true

void printColumnMajorMatrix(float *A, int nrRows, int nrCols) {
    for(int i = 0; i < nrRows; ++i){
        for(int j = 0; j < nrCols; ++j){
            std::cout << A[nrRows*j+i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

void printDeviceColumnMajorMatrix(float * dA, int nrRows, int nrCols) {
	int size = nrRows*nrCols;
	float hA[size];
        hipMemcpy(hA, dA, size*sizeof(float), hipMemcpyDeviceToHost);
        printColumnMajorMatrix(hA,nrRows,nrCols);
}

void checkError(int line, hipblasStatus_t stat) {
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		std::cout << "CUBLAS error at line: " << line << ", status: " << stat << std::endl;
		exit(EXIT_FAILURE);
        }
}

void checkCuRandError(int line, hiprandStatus_t stat) {
        if (stat != HIPRAND_STATUS_SUCCESS) {
                std::cout << "CURAND error at line: " << line << ", status: " << stat << std::endl;
                exit(EXIT_FAILURE);
        }
}

RBM::RBM(int visible, int hidden, float rate) {
	blockSize=512;
	numVisible = visible;
	numHidden = hidden;
	learningRate = rate;

	int weightsNumber = (numVisible+1)*(numHidden+1);  // because of bias
	hipMalloc(&dWeights,weightsNumber*sizeof(float));
	checkCuRandError(__LINE__,hiprandCreateGenerator(&generator,HIPRAND_RNG_QUASI_DEFAULT));
//	checkCuRandError(__LINE__,hiprandSetPseudoRandomGeneratorSeed(generator,1234ULL));
//	hiprandGenerateUniform(generator,dWeights,weightsNumber);
	checkCuRandError(__LINE__,hiprandGenerateNormal(generator,dWeights,weightsNumber,0.0,0.1));

	std::cout <<"Initial weights="<<std::endl;	
	printDeviceColumnMajorMatrix(dWeights,numVisible+1,numHidden+1);
		
	checkError(__LINE__,hipblasCreate(&handle));
	std::cout << "CUBLAS library context created" << std::endl;
}

RBM::~RBM() {
	hipFree(dWeights);
	hipblasDestroy(handle);
	hiprandDestroyGenerator(generator);
	std::cout << "CUBLAS library context destroyed" << std::endl;
}

float * RBM::hiddenActivationProbability(float * dVisibleUnitsStates, int examplesNumber) {
	float *dHiddenUnitsActivationEnergy; 		//matrix of float values of dim exh
	float *dHiddenUnitsActivationProbabilities;	//matrix of [0,1] values of dim exh
	hipMalloc(&dHiddenUnitsActivationEnergy,(numHidden+1)*examplesNumber*sizeof(float));
	hipMalloc(&dHiddenUnitsActivationProbabilities,(numHidden+1)*examplesNumber*sizeof(float));
        
	if(DEBUG) std::cout << "Calculating hidden units activation energies" << std::endl;
	
	const float alpha = 1;
        const float beta =  0;
        checkError(__LINE__,hipblasSgemm(
		handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                examplesNumber,
                numHidden+1,
                numVisible+1,
                &alpha,
                dVisibleUnitsStates,
                examplesNumber,//lda 
                dWeights,
                numVisible+1,//ldb
                &beta,
                dHiddenUnitsActivationEnergy,
                examplesNumber));//ldc

	if(DEBUG) printDeviceColumnMajorMatrix(dHiddenUnitsActivationEnergy,examplesNumber,numHidden+1);

        //calculate hidden probabilities
        int blockNumber = examplesNumber*(numHidden+1)/blockSize + 1;
        if(DEBUG) std::cout << "Calculating hidden probabilities " << std::endl;
        sigmoid<<<blockNumber,blockSize>>>(dHiddenUnitsActivationEnergy,dHiddenUnitsActivationProbabilities,examplesNumber*(numHidden+1));
        if(DEBUG) printDeviceColumnMajorMatrix(dHiddenUnitsActivationProbabilities,examplesNumber,numHidden+1);

	hipFree(dHiddenUnitsActivationEnergy);
	return dHiddenUnitsActivationProbabilities;
}

float * RBM::visibleActivationProbability(float * dHiddenUnitsStates, int examplesNumber) {
	float *dVisibleUnitsActivationEnergy;		//matrix of float values of dim exv
	float *dVisibleUnitsActivationProbabilities;	//matrix of [0,1] values of dim exv
	hipMalloc(&dVisibleUnitsActivationEnergy,(numVisible+1)*examplesNumber*sizeof(float));
	hipMalloc(&dVisibleUnitsActivationProbabilities,(numVisible+1)*examplesNumber*sizeof(float));

	if(DEBUG) std::cout << "Calculating visible units activation energies"<<std::endl;

	const float alpha = 1;
        const float beta =  0;
        checkError(__LINE__,hipblasSgemm(
        	handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_T,
                examplesNumber,
                numVisible+1,
                numHidden+1,
                &alpha,
                dHiddenUnitsStates,
                examplesNumber,//lda 
                dWeights,
                numVisible+1,//ldb
                &beta,
                dVisibleUnitsActivationEnergy,
                examplesNumber));//ldc
	if(DEBUG) printDeviceColumnMajorMatrix(dVisibleUnitsActivationEnergy,examplesNumber,numVisible+1);

        int blockNumber = examplesNumber*(numVisible+1)/blockSize + 1;
	if(DEBUG) std::cout << "Calculating visible probabilities" << blockNumber << " " << blockSize << std::endl;

        sigmoid<<<blockNumber,blockSize>>>(dVisibleUnitsActivationEnergy,dVisibleUnitsActivationProbabilities,examplesNumber*(numVisible+1));
	
	if(DEBUG) printDeviceColumnMajorMatrix(dVisibleUnitsActivationProbabilities,examplesNumber,numVisible+1);	
	
	hipFree(dVisibleUnitsActivationEnergy);
	return dVisibleUnitsActivationProbabilities;
}

float * RBM::computeAssociations(float * dVisibleUnitsActivationProbabilities, float * dHiddenUnitsActivationProbabilities, int examplesNumber) {
	float * dAssociations;	//vxh matrix
	hipMalloc(&dAssociations,(numVisible+1)*(numHidden+1)*sizeof(float));//because of bias

 	const float alpha = 1;
        const float beta =  0;
        checkError(__LINE__,hipblasSgemm(
        	handle,
                HIPBLAS_OP_T,
                HIPBLAS_OP_N,
                numVisible+1,
                numHidden+1,
                examplesNumber,
                &alpha,
                dVisibleUnitsActivationProbabilities,
                examplesNumber,//lda 
                dHiddenUnitsActivationProbabilities,
                examplesNumber,//ldb
                &beta,
                dAssociations,
                numVisible+1));//ldc
	if(DEBUG) printDeviceColumnMajorMatrix(dAssociations,numVisible+1,numHidden+1);

	return dAssociations;
}

void RBM::train(float * hTrainingData, int examplesNumber, int maxEpochs) {
	float hBias[examplesNumber];
	std::fill_n(hBias,examplesNumber,1.0);

	float *dVisibleUnitsStates; 	//device copy of training data
	float *dVisibleUnitsActivationProbabilities;//matrix of [0,1] of dim exv

	float *dHiddenUnitsStates;	//matrix of boolean values of dim exh
	float *dPositiveHiddenUnitsActivationProbabilities;//matrix of [0,1] of dim exh

        float *dNegativeHiddenUnitsActivationProbabilities;//matrix of [0,1] of dim exh

	float *dPositiveAssociations; //metrix of dimensions vxh
	float *dNegativeAssociations; //matrix of dim
	
	float * dRandom;//matrix of dim exh of random values [0,1]
	
	hipMalloc(&dVisibleUnitsStates,(numVisible+1)*examplesNumber*sizeof(float));
	hipMalloc(&dHiddenUnitsStates,(numHidden+1)*examplesNumber*sizeof(float));
	hipMalloc(&dRandom,examplesNumber*(numHidden+1)*sizeof(float));

	for(int e=0; e< maxEpochs; e++) {	        				
		//positive phase of contrastive divergence

		//copy bias to the first column
		hipMemcpy(dVisibleUnitsStates,hBias,examplesNumber*sizeof(float),hipMemcpyHostToDevice);
		//copy trainingData to remaining cells
	        hipMemcpy(&dVisibleUnitsStates[examplesNumber],hTrainingData,numVisible*examplesNumber*sizeof(float),hipMemcpyHostToDevice);

		//calculate positive hidden activation probabilities
		dPositiveHiddenUnitsActivationProbabilities = hiddenActivationProbability(dVisibleUnitsStates, examplesNumber);
	
        	if(DEBUG) std::cout << "Calculating hidden unit states" << std::endl;
        	checkCuRandError(__LINE__,hiprandGenerateUniform(generator,dRandom,examplesNumber*(numHidden+1)));
		int blockNumber = examplesNumber*(numHidden+1)/blockSize + 1;
		greaterThan<<<blockNumber,blockSize>>>(dPositiveHiddenUnitsActivationProbabilities,dRandom,dHiddenUnitsStates,examplesNumber*(numHidden+1));
        	if(DEBUG) printDeviceColumnMajorMatrix(dHiddenUnitsStates,examplesNumber,numHidden+1);
	
		if(DEBUG) std::cout << "Calculating positive associations" << std::endl;
		dPositiveAssociations = computeAssociations(dVisibleUnitsStates,dPositiveHiddenUnitsActivationProbabilities, examplesNumber);

		//negative phase of contrastive divergence

		//negative visible probabilities
		dVisibleUnitsActivationProbabilities = visibleActivationProbability(dHiddenUnitsStates,examplesNumber);
		
		//fix bias by copying ones to the first column
		if(DEBUG) std::cout << "Fixing visible units ativation probabilities" << std::endl;
                hipMemcpy(dVisibleUnitsActivationProbabilities,hBias,examplesNumber*sizeof(float),hipMemcpyHostToDevice);

		if(DEBUG) printDeviceColumnMajorMatrix(dVisibleUnitsActivationProbabilities,examplesNumber,numVisible+1);
		//negative hidden probabilities
		dNegativeHiddenUnitsActivationProbabilities = hiddenActivationProbability(dVisibleUnitsActivationProbabilities,examplesNumber);

		if(DEBUG) std::cout << "Calculating negative associations" << std::endl;
		dNegativeAssociations = computeAssociations(dVisibleUnitsActivationProbabilities,dNegativeHiddenUnitsActivationProbabilities, examplesNumber);
				
		//updating weights
		int weightsNumber = (numHidden+1)*(numVisible+1);
		blockNumber = weightsNumber/blockSize + 1;
		if(DEBUG) std::cout << "Updating weights" << std::endl;
                updateWeight<<<blockNumber,blockSize>>>(dWeights,dPositiveAssociations,dNegativeAssociations,weightsNumber,examplesNumber,learningRate);
		if(DEBUG) printDeviceColumnMajorMatrix(dWeights,numVisible+1,numHidden+1);
		
		blockNumber = examplesNumber/blockSize+1;
		//for memory eficiency we will write substraction result to the dVisibleUnitsStates
		if(DEBUG) std::cout << "subAndSquare" << std::endl;
		subAndSquare<<<blockNumber,blockSize>>>(dVisibleUnitsStates,dVisibleUnitsActivationProbabilities,(numVisible+1)*examplesNumber); 
		if(DEBUG) printDeviceColumnMajorMatrix(dVisibleUnitsStates,examplesNumber,(numVisible+1));

		blockNumber = examplesNumber/2/blockSize+1;
		if(DEBUG) std::cout << "sumReduce" << std::endl;
		sumReduce<<<blockNumber,blockSize>>>(dVisibleUnitsStates,(numVisible+1)*examplesNumber);
		if(DEBUG) printDeviceColumnMajorMatrix(dVisibleUnitsStates,examplesNumber,(numVisible+1));		

		float hError;	
		hipMemcpy(&hError,dVisibleUnitsStates,sizeof(float),hipMemcpyDeviceToHost);

		std::cout << "Error for epoch " << e << " is " << hError << std::endl;
		
		hipFree(dVisibleUnitsActivationProbabilities);
                hipFree(dPositiveHiddenUnitsActivationProbabilities);
                hipFree(dNegativeHiddenUnitsActivationProbabilities);
                hipFree(dPositiveAssociations);
                hipFree(dNegativeAssociations);	
	}

	hipFree(dRandom);
	hipFree(dVisibleUnitsStates);
	hipFree(dHiddenUnitsStates);

	std::cout << "Learned weights:" << std::endl;
	printDeviceColumnMajorMatrix(dWeights,numVisible+1,numHidden+1);
}

float * RBM::hiddenActivationProbability(float *hVisible) {
	float * dVisible;
	float * dHidden;
	float * hHidden;
	float * dRandom;

	hipMalloc(&dVisible,(numVisible+1)*sizeof(float));
	hipMalloc(&dHidden,(numHidden+1)*sizeof(float));

	float bias = 1.0;
	hipMemcpy(dVisible,&bias,sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(&dVisible[1],hVisible,numVisible*sizeof(float),hipMemcpyHostToDevice); //set biast

	dHidden = hiddenActivationProbability(dVisible, 1);

	//sample
        hipMalloc(&dRandom,(numHidden+1)*sizeof(float));
        checkCuRandError(__LINE__,hiprandGenerateUniform(generator,dRandom,numHidden+1));
	int blockSize = 512;
        int blockNumber = (numHidden+1)/blockSize + 1;
        greaterThan<<<blockNumber,blockSize>>>(dHidden,dRandom,dHidden,numHidden+1);

	hHidden = (float*) malloc(numHidden*sizeof(float));
	hipMemcpy(hHidden,&dHidden[1],numHidden*sizeof(float),hipMemcpyDeviceToHost);
	hipFree(dHidden);
	hipFree(dVisible);
	hipFree(dRandom);
	return hHidden;
}
