#include <iostream>
#include <hipblas.h>
#include <hiprand.h>
#include "rbm.h"
#include "rbm_kernels.h"

#define INFO  false
#define DEBUG false

void printColumnMajorMatrix(float *A, int nrRows, int nrCols) {
    for (int i = 0; i < nrRows; ++i) {
        for (int j = 0; j < nrCols; ++j)
            std::cout << A[nrRows * j + i] << " ";
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

void printDeviceColumnMajorMatrix(float *dA, int nrRows, int nrCols) {
    int   size = nrRows * nrCols;
    float hA[size];

    hipMemcpy(hA, dA, size * sizeof(float), hipMemcpyDeviceToHost);
    printColumnMajorMatrix(hA, nrRows, nrCols);
}

void checkCuBlasError(int line, hipblasStatus_t stat) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "CUBLAS error at line: " << line << ", status: " << stat << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCuRandError(int line, hiprandStatus_t stat) {
    if (stat != HIPRAND_STATUS_SUCCESS) {
        std::cout << "CURAND error at line: " << line << ", status: " << stat << std::endl;
        exit(EXIT_FAILURE);
    }
}

RBM::RBM(int visible, int hidden, float rate) {
    blockSize = 512;    // it is assumed that number of training examples is big (exv and exh are bigger than 512);
                        // otherwise blockSize should be set dynamically
    numVisible   = visible;
    numHidden    = hidden;
    learningRate = rate;

    int weightsNumber = (numVisible + 1) * (numHidden + 1);  // +1 because of bias
    hipMalloc(&dWeights, weightsNumber * sizeof(float));
    checkCuRandError(__LINE__, hiprandCreateGenerator(&generator, HIPRAND_RNG_QUASI_DEFAULT));
    checkCuRandError(__LINE__, hiprandGenerateNormal(generator, dWeights, weightsNumber, 0.0, 0.1));

    if (INFO) {
        std::cout << "Initial weights=" << std::endl;
        printDeviceColumnMajorMatrix(dWeights, numVisible + 1, numHidden + 1);
    }

    checkCuBlasError(__LINE__, hipblasCreate(&handle));
    std::cout << "RBM initialized" << std::endl;
}

RBM::~RBM() {
    hipFree(dWeights);
    hipblasDestroy(handle);
    hiprandDestroyGenerator(generator);
    std::cout << "RBM destroyed" << std::endl;
}

float *RBM::hiddenActivationProbabilities(float *dVisibleUnitsStates, int examplesNumber) {
    float *dHiddenUnitsActivationEnergy;        // matrix of float values of dim exh
    float *dHiddenUnitsActivationProbabilities; // matrix of [0,1] values of dim exh

    hipMalloc(&dHiddenUnitsActivationEnergy, (numHidden + 1) * examplesNumber * sizeof(float));
    hipMalloc(&dHiddenUnitsActivationProbabilities, (numHidden + 1) * examplesNumber * sizeof(float));

    if (DEBUG) std::cout << "Calculating hidden units activation energies" << std::endl;

    const float alpha = 1;
    const float beta  = 0;
    checkCuBlasError(__LINE__, hipblasSgemm(
                         handle,
                         HIPBLAS_OP_N,
                         HIPBLAS_OP_N,
                         examplesNumber,
                         numHidden + 1,
                         numVisible + 1,
                         &alpha,
                         dVisibleUnitsStates,
                         examplesNumber, // lda
                         dWeights,
                         numVisible + 1, // ldb
                         &beta,
                         dHiddenUnitsActivationEnergy,
                         examplesNumber)); // ldc

    if (DEBUG) printDeviceColumnMajorMatrix(dHiddenUnitsActivationEnergy, examplesNumber, numHidden + 1);

    int blockNumber = examplesNumber * (numHidden + 1) / blockSize + 1;
    if (DEBUG) std::cout << "Calculating hidden probabilities" << std::endl;
    sigmoid<<<blockNumber, blockSize>>>(dHiddenUnitsActivationEnergy, dHiddenUnitsActivationProbabilities, examplesNumber * (numHidden + 1));
    if (DEBUG) printDeviceColumnMajorMatrix(dHiddenUnitsActivationProbabilities, examplesNumber, numHidden + 1);

    hipFree(dHiddenUnitsActivationEnergy);
    return dHiddenUnitsActivationProbabilities;
}

float *RBM::visibleActivationProbabilities(float *dHiddenUnitsStates, int examplesNumber) {
    float *dVisibleUnitsActivationEnergy;        // matrix of float values of dim exv
    float *dVisibleUnitsActivationProbabilities; // matrix of [0,1] values of dim exv

    hipMalloc(&dVisibleUnitsActivationEnergy, (numVisible + 1) * examplesNumber * sizeof(float));
    hipMalloc(&dVisibleUnitsActivationProbabilities, (numVisible + 1) * examplesNumber * sizeof(float));

    if (DEBUG) std::cout << "Calculating visible units activation energies" << std::endl;

    const float alpha = 1;
    const float beta  = 0;
    checkCuBlasError(__LINE__, hipblasSgemm(
                         handle,
                         HIPBLAS_OP_N,
                         HIPBLAS_OP_T,
                         examplesNumber,
                         numVisible + 1,
                         numHidden + 1,
                         &alpha,
                         dHiddenUnitsStates,
                         examplesNumber, // lda
                         dWeights,
                         numVisible + 1, // ldb
                         &beta,
                         dVisibleUnitsActivationEnergy,
                         examplesNumber)); // ldc

    if (DEBUG) printDeviceColumnMajorMatrix(dVisibleUnitsActivationEnergy, examplesNumber, numVisible + 1);

    int blockNumber = examplesNumber * (numVisible + 1) / blockSize + 1;
    if (DEBUG) std::cout << "Calculating visible probabilities" << blockNumber << " " << blockSize << std::endl;

    sigmoid<<<blockNumber, blockSize>>>(dVisibleUnitsActivationEnergy, dVisibleUnitsActivationProbabilities, examplesNumber * (numVisible + 1));

    if (DEBUG) printDeviceColumnMajorMatrix(dVisibleUnitsActivationProbabilities, examplesNumber, numVisible + 1);

    hipFree(dVisibleUnitsActivationEnergy);
    return dVisibleUnitsActivationProbabilities;
}

float *RBM::computeAssociations(float *dVisibleUnitsActivationProbabilities,
                                float *dHiddenUnitsActivationProbabilities,
                                int    examplesNumber) {
    float *dAssociations; // vxh matrix

    hipMalloc(&dAssociations, (numVisible + 1) * (numHidden + 1) * sizeof(float)); // +1 because of bias

    const float alpha = 1;
    const float beta  = 0;
    checkCuBlasError(__LINE__, hipblasSgemm(
                         handle,
                         HIPBLAS_OP_T,
                         HIPBLAS_OP_N,
                         numVisible + 1,
                         numHidden + 1,
                         examplesNumber,
                         &alpha,
                         dVisibleUnitsActivationProbabilities,
                         examplesNumber, // lda
                         dHiddenUnitsActivationProbabilities,
                         examplesNumber, // ldb
                         &beta,
                         dAssociations,
                         numVisible + 1)); // ldc

    if (DEBUG) printDeviceColumnMajorMatrix(dAssociations, numVisible + 1, numHidden + 1);

    return dAssociations;
}

// a contrastive divergence (CD_1) learning algorithm; batched version
void RBM::train(float *hTrainingData, int examplesNumber, int maxEpochs) {
    float hBias[examplesNumber];                        // will be added as a first column of training data
    std::fill_n(hBias, examplesNumber, 1.0);

    float *dVisibleUnitsStates;                         // device copy of training data
    float *dVisibleUnitsActivationProbabilities;        // matrix of [0,1] of dimensions exv

    float *dHiddenUnitsStates;                          // matrix of boolean values of dimensions exh
    float *dPositiveHiddenUnitsActivationProbabilities; // matrix of [0,1] of dimensions exh

    float *dNegativeHiddenUnitsActivationProbabilities; // matrix of [0,1] of dimensions exh

    float *dPositiveAssociations;                       // matrix of dimensions vxh
    float *dNegativeAssociations;                       // matrix of dimensions vxh

    float *dRandom;                                     // matrix of dimensions exh of random values [0,1]

    hipMalloc(&dVisibleUnitsStates, (numVisible + 1) * examplesNumber * sizeof(float));
    hipMalloc(&dHiddenUnitsStates, (numHidden + 1) * examplesNumber * sizeof(float));
    hipMalloc(&dRandom, examplesNumber * (numHidden + 1) * sizeof(float));

    for (int e = 0; e < maxEpochs; e++) {
        // a positive phase of the contrastive divergence
        if (DEBUG) std::cout << "Epoch " << e << std::endl;
        // copy bias to the first column
        hipMemcpy(dVisibleUnitsStates, hBias, examplesNumber * sizeof(float), hipMemcpyHostToDevice);

        // copy training data to remaining cells
        hipMemcpy(&dVisibleUnitsStates[examplesNumber],
                   hTrainingData,
                   numVisible * examplesNumber * sizeof(float),
                   hipMemcpyHostToDevice);

        // calculate positive hidden activation probabilities
        dPositiveHiddenUnitsActivationProbabilities = hiddenActivationProbabilities(dVisibleUnitsStates, examplesNumber);

        if (DEBUG) std::cout << "Calculating hidden unit states by sampling" << std::endl;
        checkCuRandError(__LINE__, hiprandGenerateUniform(generator, dRandom, examplesNumber * (numHidden + 1)));
        int blockNumber = examplesNumber * (numHidden + 1) / blockSize + 1;
        greaterThan<<<blockNumber, blockSize>>>(dPositiveHiddenUnitsActivationProbabilities, dRandom, dHiddenUnitsStates, examplesNumber * (numHidden + 1));
        if (DEBUG) printDeviceColumnMajorMatrix(dHiddenUnitsStates, examplesNumber, numHidden + 1);

        if (DEBUG) std::cout << "Calculating positive associations" << std::endl;
        dPositiveAssociations = computeAssociations(dVisibleUnitsStates,
                                                    dPositiveHiddenUnitsActivationProbabilities,
                                                    examplesNumber);

        // a negative (reconstruction) phase of the contrastive divergence

        // calculate negative visible probabilities
        dVisibleUnitsActivationProbabilities = visibleActivationProbabilities(dHiddenUnitsStates, examplesNumber);

        if (DEBUG) std::cout << "Fixing visible units activation probabilities by setting bias to the first column" << std::endl;
        hipMemcpy(dVisibleUnitsActivationProbabilities, hBias, examplesNumber * sizeof(float), hipMemcpyHostToDevice);

        if (DEBUG) printDeviceColumnMajorMatrix(dVisibleUnitsActivationProbabilities, examplesNumber, numVisible + 1);
        // negative hidden probabilities
        dNegativeHiddenUnitsActivationProbabilities = hiddenActivationProbabilities(
            dVisibleUnitsActivationProbabilities,
            examplesNumber);

        if (DEBUG) std::cout << "Calculating negative associations" << std::endl;
        dNegativeAssociations = computeAssociations(dVisibleUnitsActivationProbabilities,
                                                    dNegativeHiddenUnitsActivationProbabilities,
                                                    examplesNumber);

        if (DEBUG) std::cout << "Updating weights" << std::endl;
        int weightsNumber = (numHidden + 1) * (numVisible + 1);
        blockNumber = weightsNumber / blockSize + 1;
        updateWeight<<<blockNumber, blockSize>>>(dWeights, dPositiveAssociations, dNegativeAssociations, weightsNumber, examplesNumber, learningRate);
        if (DEBUG) printDeviceColumnMajorMatrix(dWeights, numVisible + 1, numHidden + 1);

        if (DEBUG) std::cout << "Calculating error - squares of subtractions" << std::endl;
        blockNumber = examplesNumber / blockSize + 1;
        // for memory efficiency we will write subtraction result to one of the input matrices (dVisibleUnitsStates)
        subAndSquare<<<blockNumber, blockSize>>>(dVisibleUnitsStates, dVisibleUnitsActivationProbabilities, (numVisible + 1) * examplesNumber);
        if (DEBUG) printDeviceColumnMajorMatrix(dVisibleUnitsStates, examplesNumber, (numVisible + 1));

        blockNumber = examplesNumber / 2 / blockSize + 1;
        if (DEBUG) std::cout << "Calculation error - reducing sum" << std::endl;
        sumReduce<<<blockNumber, blockSize>>>(dVisibleUnitsStates, (numVisible + 1) * examplesNumber);
        if (DEBUG) printDeviceColumnMajorMatrix(dVisibleUnitsStates, examplesNumber, (numVisible + 1));

        float hError;
        hipMemcpy(&hError, dVisibleUnitsStates, sizeof(float), hipMemcpyDeviceToHost);

        std::cout << "Error after epoch " << e + 1 << " is " << hError << std::endl;

        hipFree(dVisibleUnitsActivationProbabilities);
        hipFree(dPositiveHiddenUnitsActivationProbabilities);
        hipFree(dNegativeHiddenUnitsActivationProbabilities);
        hipFree(dPositiveAssociations);
        hipFree(dNegativeAssociations);
    }

    hipFree(dRandom);
    hipFree(dVisibleUnitsStates);
    hipFree(dHiddenUnitsStates);

    if (INFO) std::cout << "Learned weights:" << std::endl;
    if (INFO) printDeviceColumnMajorMatrix(dWeights, numVisible + 1, numHidden + 1);
}

float *RBM::hiddenStates(float *hVisible) {
    float *dVisible;
    float *dHidden;
    float *hHidden;
    float *dRandom;

    hipMalloc(&dVisible, (numVisible + 1) * sizeof(float));
    hipMalloc(&dHidden, (numHidden + 1) * sizeof(float));

    float bias = 1.0;
    hipMemcpy(dVisible, &bias, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dVisible[1], hVisible, numVisible * sizeof(float), hipMemcpyHostToDevice); // set bias

    dHidden = hiddenActivationProbabilities(dVisible, 1);

    // sampling
    hipMalloc(&dRandom, (numHidden + 1) * sizeof(float));
    checkCuRandError(__LINE__, hiprandGenerateUniform(generator, dRandom, numHidden + 1));
    int blockNumber = (numHidden + 1) / blockSize + 1;
    greaterThan<<<blockNumber, blockSize>>>(dHidden, dRandom, dHidden, numHidden + 1);

    hHidden = (float *) malloc(numHidden * sizeof(float));
    hipMemcpy(hHidden, &dHidden[1], numHidden * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dHidden);
    hipFree(dVisible);
    hipFree(dRandom);
    return hHidden;
}
